#include "ndt_gpu/MatrixDevice.h"
#include "ndt_gpu/debug.h"

namespace gpu {
template <typename T>
void MatrixDevice<T>::memAlloc()
{
	if (buffer_ != NULL && fr_) {
		checkCudaErrors(hipFree(buffer_));
		buffer_ = NULL;
	}

	checkCudaErrors(hipMalloc(&buffer_, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipMemset(buffer_, 0, sizeof(double) * rows_ * cols_ * offset_));
	checkCudaErrors(hipDeviceSynchronize());
	fr_ = true;
}

template <typename T>
void MatrixDevice<T>::memFree()
{
	if (fr_) {
		if (buffer_ != NULL) {
			checkCudaErrors(hipFree(buffer_));
			buffer_ = NULL;
		}
	}
}

template <typename T>
SquareMatrixDevice<T>::SquareMatrixDevice(int size) :
	MatrixDevice(size, size)
{}

}
