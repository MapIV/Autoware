#include "hip/hip_runtime.h"
#include "include/euclidean_cluster.h"
#include "include/utilities.h"
#include "include/voxel_grid.h"
#include <hip/hip_runtime.h>

#define TEST_VERTEX_ 1

__global__ void frontierInitialize2(int *frontier_array, int point_num)
{
	for (int i = threadIdx.x + blockIdx.x * blockDim.x; i < point_num; i += blockDim.x * gridDim.x) {
		frontier_array[i] = 1;
	}
}

__global__ void vertexBasedClustering(int *adjacent_list_loc, int *adjacent_list, int point_num, int *cluster_name, int *frontier_array1, int *frontier_array2, bool *changed)
{
	__shared__ bool schanged;

	if (threadIdx.x == 0)
		schanged = false;
	__syncthreads();

	for (int pid = threadIdx.x + blockIdx.x * blockDim.x; pid < point_num; pid += blockDim.x * gridDim.x) {
		if (frontier_array1[pid] == 1) {
			frontier_array1[pid] = 0;
			int cname = cluster_name[pid];
			bool c = false;
			int start = adjacent_list_loc[pid];
			int end = adjacent_list_loc[pid + 1];

			// Iterate through neighbors' ids
			for (int i = start; i < end; i++) {
				int nid = adjacent_list[i];
				int nname = cluster_name[nid];
				if (cname < nname) {
					atomicMin(cluster_name + nid, cname);
					frontier_array2[nid] = 1;
					schanged = true;
					//*changed = true;
				} else if (cname > nname) {
					cname = nname;
					c = true;
				}
			}

			if (c) {
				atomicMin(cluster_name + pid, cname);
				frontier_array2[pid] = 1;
				schanged = true;
				//*changed = true;
			}
		}
	}
	__syncthreads();

	if (threadIdx.x == 0 && schanged)
		*changed = true;
}

/* Iterate through the list of remaining clusters and mark the corresponding
 * location on cluster location array by 1
 */
__global__ void clusterMark3(int *cluster_list, int *cluster_location, int cluster_num)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;

	for (int i = idx; i < cluster_num; i += blockDim.x * gridDim.x) {
		cluster_location[cluster_list[i]] = 1;
	}
}


void GpuEuclideanCluster2::extractClusters6()
{
	long long total_time, build_graph, clustering_time;
	int iteration_num;

	extractClusters6(total_time, build_graph, clustering_time, iteration_num);
}



void GpuEuclideanCluster2::extractClusters6(long long &total_time, long long &build_graph, long long &clustering_time, int &iteration_num)
{
#ifdef DEBUG_
	std::cout << "VERTEX-BASED 2: Use octree" << std::endl;
#endif
	total_time = build_graph = clustering_time = 0;
	iteration_num = 0;

	struct timeval start, end;

	gettimeofday(&start, NULL);


	initClusters();

	gettimeofday(&end, NULL);

	total_time += timeDiff(start, end);

	gettimeofday(&start, NULL);
	GVoxelGrid new_grid(x_, y_, z_, point_num_, threshold_, threshold_, threshold_);
	gettimeofday(&end, NULL);

#ifdef DEBUG_
	std::cout << "Create voxel = " << timeDiff(start, end) << std::endl;
#endif

	build_graph += timeDiff(start, end);
	total_time += timeDiff(start, end);

	int *adjacent_count = NULL;
	int *adjacent_list = NULL;
	int adjacent_list_size = 0;

	gettimeofday(&start, NULL);
	new_grid.createAdjacentList(&adjacent_count, &adjacent_list, &adjacent_list_size, threshold_);

	if (adjacent_list_size == 0) {
		cluster_num_ = point_num_;

		return;
	}

	int block_x = (point_num_ < block_size_x_) ? point_num_ : block_size_x_;
	int grid_x = (point_num_ - 1) / block_x + 1;

	gettimeofday(&end, NULL);

#ifdef DEBUG_
	std::cout << "Create adjacency list = " << timeDiff(start, end) << std::endl;
#endif

	build_graph += timeDiff(start, end);
	total_time += timeDiff(start, end);

#ifdef DEBUG_
	std::cout << "Build graph = " << timeDiff(start, end) << std::endl;
#endif

	bool *changed;

	bool hchanged;
	checkCudaErrors(hipMalloc(&changed, sizeof(bool)));

	int *frontier_array1, *frontier_array2;

	gettimeofday(&start, NULL);

	checkCudaErrors(hipMalloc(&frontier_array1, sizeof(int) * point_num_));
	checkCudaErrors(hipMalloc(&frontier_array2, sizeof(int) * point_num_));

	frontierInitialize2<<<grid_x, block_x>>>(frontier_array1, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipMemset(frontier_array2, 0, sizeof(int) * point_num_));
	checkCudaErrors(hipDeviceSynchronize());

	int itr = 0;

	do {
		hchanged = false;
		checkCudaErrors(hipMemcpy(changed, &hchanged, sizeof(bool), hipMemcpyHostToDevice));

		vertexBasedClustering<<<grid_x, block_x>>>(adjacent_count, adjacent_list, point_num_, cluster_name_, frontier_array1, frontier_array2, changed);
		checkCudaErrors(hipGetLastError());
		checkCudaErrors(hipDeviceSynchronize());

		int *tmp;

		tmp = frontier_array1;
		frontier_array1 = frontier_array2;
		frontier_array2 = tmp;

		checkCudaErrors(hipMemcpy(&hchanged, changed, sizeof(bool), hipMemcpyDeviceToHost));

		itr++;
	} while (hchanged);



	gettimeofday(&end, NULL);

	clustering_time += timeDiff(start, end);
	total_time += timeDiff(start, end);

#ifdef DEBUG_
	std::cout << "Iteration = " << timeDiff(start, end) << " itr_num = " << itr << std::endl;
#endif

	iteration_num = itr;


	// renaming clusters
	int *cluster_location;

	gettimeofday(&start, NULL);
	checkCudaErrors(hipMalloc(&cluster_location, sizeof(int) * (point_num_ + 1)));
	checkCudaErrors(hipMemset(cluster_location, 0, sizeof(int) * (point_num_ + 1)));

	clusterMark3<<<grid_x, block_x>>>(cluster_name_, cluster_location, point_num_);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	GUtilities::exclusiveScan(cluster_location, point_num_ + 1, &cluster_num_);

	renamingClusters(cluster_name_, cluster_location, point_num_);

	checkCudaErrors(hipFree(adjacent_count));
	checkCudaErrors(hipFree(adjacent_list));
	checkCudaErrors(hipFree(frontier_array1));
	checkCudaErrors(hipFree(frontier_array2));
	checkCudaErrors(hipFree(changed));
	checkCudaErrors(hipFree(cluster_location));
	gettimeofday(&end, NULL);

	total_time += timeDiff(start, end);

#ifndef DEBUG_
	std::cout << "FINAL CLUSTER NUM = " << cluster_num_ << std::endl << std::endl;
#endif
}
